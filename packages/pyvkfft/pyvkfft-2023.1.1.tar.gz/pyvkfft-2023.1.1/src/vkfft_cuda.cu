#include "hip/hip_runtime.h"
/* PyVkFFT
   (c) 2021- : ESRF-European Synchrotron Radiation Facility
       authors:
         Vincent Favre-Nicolin, favre@esrf.fr
*/

// We use the CUDA backend
#define VKFFT_BACKEND 1

#include <iostream>
#include <fstream>
#include <memory>
using namespace std;
#include "vkFFT.h"
typedef float2 Complex;

#ifdef _WIN32
#define LIBRARY_API extern "C" __declspec(dllexport)
#else
#define LIBRARY_API extern "C"
#endif


LIBRARY_API VkFFTConfiguration* make_config(const size_t, const size_t, const size_t, const size_t, void*, void*, void*,
                                const int, const size_t, const int, const int, const int, const int,
                                const int, const int, const size_t, const int, const int, const int);

LIBRARY_API VkFFTApplication* init_app(const VkFFTConfiguration*, int*);

LIBRARY_API int fft(VkFFTApplication* app, void*, void*);

LIBRARY_API int ifft(VkFFTApplication* app, void*, void*);

LIBRARY_API void free_app(VkFFTApplication* app);

LIBRARY_API void free_config(VkFFTConfiguration *config);

LIBRARY_API uint32_t vkfft_version();

LIBRARY_API int cuda_runtime_version();

LIBRARY_API int cuda_driver_version();

LIBRARY_API int cuda_compile_version();


class PyVkFFT
{
  public:
    PyVkFFT(const int nx, const int ny, const int nz, const int fftdim, void* hstream,
            const int norm, const int precision, const int r2c)
    {

    };
  private:
    VkFFTConfiguration mConf;
    VkFFTApplication mApp;
    VkFFTApplication mLaunchParams;
};


/** Create the VkFFTConfiguration from the array parameters
*
* \param nx, ny, nz: dimensions of the array. The fast axis is x. In the corresponding numpy array,
* this corresponds to a shape of (nz, ny, nx)
* \param fftdim: the dimension of the transform. If nz>1 and fftdim=2, the transform is only made
* on the x and y axes
* \param buffer, buffer_out: pointer to the GPU data source and destination arrays. These
*  can be fake and the actual buffers supplied in fft() and ifft. However buffer should be non-zero,
*  and buffer_out should be non-zero only for an out-of-place transform.
* \param hstream: the stream handle (hipStream_t)
* \param norm: 0, the L2 norm is multiplied by the size on each transform, 1, the inverse transform
*   divides the L2 norm by the size.
* \param precision: number of bits per float, 16=half, 32=single, 64=double precision
* \return: the pointer to the newly created VkFFTConfiguration, or 0 if an error occurred.
*/
VkFFTConfiguration* make_config(const size_t nx, const size_t ny, const size_t nz, const size_t fftdim,
                                void *buffer, void *buffer_out, void* hstream,
                                const int norm, const size_t precision, const int r2c, const int dct,
                                const int disableReorderFourStep, const int registerBoost,
                                const int useLUT, const int keepShaderCode, const size_t n_batch,
                                const int skipx, const int skipy, const int skipz)
{
  VkFFTConfiguration *config = new VkFFTConfiguration({});
  config->FFTdim = fftdim;
  config->size[0] = nx;
  config->size[1] = ny;
  config->size[2] = nz;
  config->numberBatches = n_batch;

  config->omitDimension[0] = skipx;
  config->omitDimension[1] = skipy;
  config->omitDimension[2] = skipz;

  config->normalize = norm;
  config->performR2C = r2c;
  config->performDCT = dct;

  if(disableReorderFourStep>=0)
    config->disableReorderFourStep = disableReorderFourStep;

  if(registerBoost>=0)
    config->registerBoost = registerBoost;

  if(useLUT>=0)
    config->useLUT = useLUT;

  if(keepShaderCode>=0)
    config->keepShaderCode = keepShaderCode;

  switch(precision)
  {
      case 2 : config->halfPrecision = 1;
      case 8 : config->doublePrecision = 1;
  };

  hipDevice_t *dev = new hipDevice_t;
  if(hstream != 0)
  {
    // Get context then device from current context
    hipCtx_t ctx = nullptr;
    hipError_t res = cuStreamGetCtx ((hipStream_t)hstream, &ctx);
    if(res != hipSuccess)
    {
      cout << "Could not get the current device from given stream"<<endl;
      return 0;
    }
    res = hipCtxPushCurrent (ctx);
    res = hipCtxGetDevice(dev);
    if(res != hipSuccess)
    {
      cout << "Could not get the current device from supplied stream's context."<<endl;
      return 0;
    }
    res = hipCtxPopCurrent (&ctx);

    config->stream = new hipStream_t((hipStream_t) hstream);
    config->num_streams = 1;
  }
  else
  {
    // Get device from current context
    hipError_t res = hipCtxGetDevice(dev);
    if(res != hipSuccess)
    {
      cout << "Could not get the current device. Was a CUDA context created ?"<<endl;
      return 0;
    }
  }
  config->device = dev;

  void ** pbuf = new void*;
  *pbuf = buffer;

  uint64_t* psize = new uint64_t;
  uint64_t* psizein = psize;

  if(r2c)
  {
    *psize = (uint64_t)((nx / 2 + 1) * ny * nz * precision * (size_t)2);
    if(buffer_out != NULL)
    {
      psizein = new uint64_t;
      *psizein = (uint64_t)(nx * ny * nz * precision);
      config->inverseReturnToInputBuffer = 1;
			config->inputBufferStride[0] = nx;
			config->inputBufferStride[1] = nx * ny;
			config->inputBufferStride[2] = nx * ny * nz;
    }
  }
  else
  {
    if(dct) *psize = (uint64_t)(nx * ny * nz * precision);
    else *psize = (uint64_t)(nx * ny * nz * precision * (size_t)2);
  }

  config->bufferSize = psize;

  if(buffer_out != NULL)
  {
    // Calculations are made in buffer, so with buffer != inputBuffer we keep the original data
    void ** pbufout = new void*;
    *pbufout = buffer_out;

    config->buffer = pbufout;
    config->inputBuffer = pbuf;

    config->inputBufferSize = psizein;

    config->isInputFormatted = 1;
  }
  else
  {
    config->buffer = pbuf;
  }

  /*
  cout << "make_config: "<<config<<" "<<endl<< config->buffer<<", "<< *(config->buffer)<<", "
       << config->size[0] << " " << config->size[1] << " " << config->size[2] << " "<< config->FFTdim
       << " " << *(config->bufferSize) << endl;
  */
  return config;
}

/** Initialise the VkFFTApplication from the given configuration.
*
* \param config: the pointer to the VkFFTConfiguration
* \return: the pointer to the newly created VkFFTApplication
*/
VkFFTApplication* init_app(const VkFFTConfiguration* config, int *res)
{
  VkFFTApplication* app = new VkFFTApplication({});
  *res = initializeVkFFT(app, *config);
  /*
  cout << "init_app: "<<config<<endl<< config->buffer<<", "<< *(config->buffer)<<", "
       << config->size[0] << " " << config->size[1] << " " << config->size[2] << " "<< config->FFTdim
       << " " << *(config->bufferSize) << endl<<endl;
  cout<<res<<endl<<endl;
  */
  if(*res!=0)
  {
    delete app;
    return 0;
  }
  return app;
}

int fft(VkFFTApplication* app, void *in, void *out)
{
  // Modify the original app only to avoid allocating
  // new buffer pointers in memory
  *(app->configuration.buffer) = out;
  *(app->configuration.inputBuffer) = in;
  *(app->configuration.outputBuffer) = out;

  VkFFTLaunchParams par = {};
  par.buffer =  app->configuration.buffer;
  par.inputBuffer = app->configuration.inputBuffer;
  par.outputBuffer = app->configuration.outputBuffer;

  return VkFFTAppend(app, -1, &par);
}

int ifft(VkFFTApplication* app, void *in, void *out)
{
  // Modify the original app only to avoid allocating
  // new buffer pointers in memory
  *(app->configuration.buffer) = out;
  *(app->configuration.inputBuffer) = in;
  *(app->configuration.outputBuffer) = out;

  VkFFTLaunchParams par = {};
  par.buffer =  app->configuration.buffer;
  par.inputBuffer = app->configuration.inputBuffer;
  par.outputBuffer = app->configuration.outputBuffer;

  return VkFFTAppend(app, 1, &par);
}

/** Free memory allocated during make_config()
*
*/
void free_app(VkFFTApplication* app)
{
  if(app != NULL)
  {
    deleteVkFFT(app);
    free(app);
  }
}

/** Free memory associated to the vkFFT app
*
*/
void free_config(VkFFTConfiguration *config)
{
  free(config->device);
  // Only frees the pointer to the buffer pointer, not the buffer itself.
  free(config->buffer);
  free(config->bufferSize);

  if((config->outputBuffer != NULL) && (config->buffer != config->outputBuffer)) free(config->outputBuffer);
  if((config->inputBuffer != NULL) && (config->buffer != config->inputBuffer)
     && (config->outputBuffer != config->inputBuffer)) free(config->inputBuffer);

  if((config->inputBufferSize != NULL) && (config->inputBufferSize != config->bufferSize))
    free(config->inputBufferSize);
  if((config->outputBufferSize != NULL) && (config->outputBufferSize != config->bufferSize)
     && (config->outputBufferSize != config->inputBufferSize)) free(config->outputBufferSize);

  if(config->stream != 0) free(config->stream);
  free(config);
}

/// Get VkFFT version
uint32_t vkfft_version()
{
  return VkFFTGetVersion();
};

/// CUDA runtime version
int cuda_runtime_version()
{
  int v=0;
  const hipError_t err = hipRuntimeGetVersion(&v);
  if(err==hipSuccess) return v;
  return 0;
};

/// CUDA driver version
int cuda_driver_version()
{
  int v=0;
  const hipError_t err = hipDriverGetVersion(&v);
  if(err==hipSuccess) return v;
  return 0;
};

/// CUDA version against which pyvkfft was compiled
int cuda_compile_version()
{
  return (int)CUDA_VERSION;
};
